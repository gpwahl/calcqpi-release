#include "hip/hip_runtime.h"
/*
This file is part of calcQPI.
calcQPI is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published
by the Free Software Foundation, either version 3 of the License,
or (at your option) any later version.
calcQPI is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
See the GNU General Public License for more details.
You should have received a copy of the GNU General Public License along with calcQPI.
If not, see <https://www.gnu.org/licenses/>.
*/

#include "mpidefs.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#ifdef _mpi_version
#include <mpi.h>
#endif

#include "qpicudakernel.h"
#include <iomanip>

#include "idl.h"

using namespace std;

__device__ __inline__ dblcomplex operator+(dblcomplex a,dblcomplex b) { return {a.real+b.real,a.imag+b.imag}; }
__device__ __inline__ dblcomplex operator*(dblcomplex a,dblcomplex b) { return {a.real*b.real-a.imag*b.imag,a.real*b.imag+a.imag*b.real}; }

__device__ __inline__ dblcomplex operator*(const gpufloat a,const dblcomplex b) { return (dblcomplex){a*b.real,a*b.imag}; }

__device__ __inline__ dblcomplex cmpexp(const dblcomplex x) { gpufloat prod=exp(x.real); return (dblcomplex){prod*cos(x.imag),prod*sin(x.imag)}; }

__device__ __inline__ void matrixmultiplication(const dblcomplex *a, const dblcomplex *b, dblcomplex *c, const gpuuint n)
{
  for(gpuuint i=0;i<n*n;i++)
    *(c+i)={0.0f,0.0f};
  for(gpuuint i=0;i<n;i++)
    for(gpuuint j=0;j<n;j++)
      for(gpuuint k=0;k<n;k++)
	*(c+IDX2C(i,j,n))=*(c+IDX2C(i,j,n))+(*(a+IDX2C(i,k,n))*(*(b+IDX2C(k,j,n))));
}

//calculate LDOS using Wannier functions - using cached functions
//g0 - two-dimensional array of complex matrices (kpointsxkpointsxbandsxbands)
//ldos - two dimensioanl array of double values (nxn)
//scat - matrix (bandsxbands)
//wf - array of two-dimensional arrays containing orbitals (bandsx((2*windows+1)*oversamp)x((2*windows+1)*oversamp))
//kpoints - number of kpoints, as specified in g0
//n - number of lattice sites in real-space lattice
//bands - number of bands
//oversamp - oversampling of real-space lattice
//window - window used for orbitals
//maxband - band up to which the orbitals are included (e.g. for superconducting calculation)
__global__ void gpucalcwannierldos(const dblcomplex *g0, gpufloat *ldos,const dblcomplex *scat,const gpufloat *wf,dblcomplex *globalbuffer,dblcomplex *globalcldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint oversamp,gpuuint window,gpuuint maxband) {
  gpuuint wanniern=oversamp*n;
  gpuuint n2=(n>>1);
  gpuuint wfsize=(2*window+1)*oversamp;
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  //sum over lattice
  dblcomplex *buffer=globalbuffer+bands*bands*globalindex,
    *cldos=globalcldos+bands*bands*globalindex;
  gpuint ipos=(gpuint)i-n2, jpos=(gpuint)j-n2;
  for(gpuuint ii=0;ii<oversamp;ii++)
    for(gpuuint jj=0;jj<oversamp;jj++)
      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=0.0f;
  //R-loop over nearest neighbours
  for(gpuint nni1=-window;nni1<=(gpuint)window;nni1++)
    for(gpuint nnj1=-window;nnj1<=(gpuint)window;nnj1++) {
      //calculate TG0(0,R)
      matrixmultiplication(scat,g0+IDX4C((kpoints+kpoints-(ipos+nni1))%kpoints,(kpoints+kpoints-(jpos+nnj1))%kpoints,0,0,kpoints,bands),buffer,bands);
      //R'-loop over nearest neighbours
      for(gpuint nni2=-window;nni2<=(gpuint)window;nni2++)
	for(gpuint nnj2=-window;nnj2<=(gpuint)window;nnj2++) {
	  //calculate G0(R',0)TG0(0,R)    
	    matrixmultiplication(g0+IDX4C((kpoints+ipos+nni2)%kpoints,(kpoints+jpos+nnj2)%kpoints,0,0,kpoints,bands),buffer,cldos,bands);	      
	  //sum over sub-unit cell sampling
	  //gpuuint x=ipos*oversamp+ii,y=jpos*oversamp+jj;
	  for(gpuuint ii=0;ii<oversamp;ii++)
	    for(gpuuint jj=0;jj<oversamp;jj++) {
	      gpuuint x=ii+window*oversamp,y=jj+window*oversamp;
	      gpufloat val=*(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern));
	      for(gpuuint o1=0;o1<maxband;o1++)
		for(gpuuint o2=0;o2<maxband;o2++) {
		  gpufloat factor=*(wf+IDX3C(o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  val-=(cldos+IDX2C(o2,o1,bands))->imag*factor;
		  val-=(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,o2,o1,kpoints,bands))->imag*factor;
		}
	      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=val;
	    }
	}
    }
}

//calculate LDOS using Wannier functions - using cached functions
//g0 - two-dimensional array of complex matrices (kpointsxkpointsxbandsxbands)
//ldos - two dimensioanl array of double values (nxn)
//scat - matrix (bandsxbands)
//wf - array of two-dimensional arrays containing orbitals (bandsx((2*windows+1)*oversamp)x((2*windows+1)*oversamp))
//kpoints - number of kpoints, as specified in g0
//n - number of lattice sites in real-space lattice
//bands - number of bands
//oversamp - oversampling of real-space lattice
//window - window used for orbitals
//maxband - band up to which the orbitals are included (e.g. for superconducting calculation)
//spin - spin-polarized calculation
__global__ void gpucalcwannierldosspin(const dblcomplex *g0, gpufloat *ldos,const dblcomplex *scat,const gpufloat *wf,dblcomplex *globalbuffer,dblcomplex *globalcldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint oversamp,gpuuint window,gpuuint maxband) {
  gpuuint wanniern=oversamp*n;
  gpuuint n2=(n>>1);
  gpuuint wfsize=(2*window+1)*oversamp;
  gpuuint spinbands=maxband>>1;
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  //sum over lattice
  dblcomplex *buffer=globalbuffer+bands*bands*globalindex,
    *cldos=globalcldos+bands*bands*globalindex;
  gpuint ipos=(gpuint)i-n2, jpos=(gpuint)j-n2;
  for(gpuuint ii=0;ii<oversamp;ii++)
    for(gpuuint jj=0;jj<oversamp;jj++)
      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=0.0f;
  //R-loop over nearest neighbours
  for(gpuint nni1=-window;nni1<=(gpuint)window;nni1++)
    for(gpuint nnj1=-window;nnj1<=(gpuint)window;nnj1++) {
      //calculate TG0(0,R)
      matrixmultiplication(scat,g0+IDX4C((kpoints+kpoints-(ipos+nni1))%kpoints,(kpoints+kpoints-(jpos+nnj1))%kpoints,0,0,kpoints,bands),buffer,bands);
      //R'-loop over nearest neighbours
      for(gpuint nni2=-window;nni2<=(gpuint)window;nni2++)
	for(gpuint nnj2=-window;nnj2<=(gpuint)window;nnj2++) {
	  //calculate G0(R',0)TG0(0,R)    
	    matrixmultiplication(g0+IDX4C((kpoints+ipos+nni2)%kpoints,(kpoints+jpos+nnj2)%kpoints,0,0,kpoints,bands),buffer,cldos,bands);
	  //gpuuint x=ipos*oversamp+ii,y=jpos*oversamp+jj;
	  //sum over sub-unit cell sampling
	  for(gpuuint ii=0;ii<oversamp;ii++)
	    for(gpuuint jj=0;jj<oversamp;jj++) {
	      gpuuint x=ii+window*oversamp,y=jj+window*oversamp;
	      gpufloat val=*(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern));
	      for(gpuuint o1=0;o1<spinbands;o1++)
		for(gpuuint o2=0;o2<spinbands;o2++) {
		  gpufloat factor=*(wf+IDX3C(o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  val-=(cldos+IDX2C(o2,o1,bands))->imag*factor;
		  val-=(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,o2,o1,kpoints,bands))->imag*factor;
		  factor=*(wf+IDX3C(spinbands+o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(spinbands+o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  val-=(cldos+IDX2C(spinbands+o2,spinbands+o1,bands))->imag*factor;
		  val-=(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,spinbands+o2,spinbands+o1,kpoints,bands))->imag*factor;
		}
	      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=val;
	    }
	}
    }
}

//calculate LDOS using Wannier functions - using cached functions
//g0 - two-dimensional array of complex matrices (kpointsxkpointsxbandsxbands)
//ldos - two dimensioanl array of double values (nxn)
//scat - matrix (bandsxbands)
//wf - array of two-dimensional arrays containing orbitals (bandsx((2*windows+1)*oversamp)x((2*windows+1)*oversamp))
//kpoints - number of kpoints, as specified in g0
//n - number of lattice sites in real-space lattice
//bands - number of bands
//oversamp - oversampling of real-space lattice
//window - window used for orbitals
//maxband - band up to which the orbitals are included (e.g. for superconducting calculation)
__global__ void gpucalcwannierjosephson(const dblcomplex *g0, gpufloat *ldos,const dblcomplex *scat,const gpufloat *wf,dblcomplex *globalbuffer,dblcomplex *globalcldos,const dblcomplex tip,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint oversamp,gpuuint window,gpuuint maxband) {
  gpuuint wanniern=oversamp*n;
  gpuuint n2=(n>>1);
  gpuuint wfsize=(2*window+1)*oversamp;
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  //sum over lattice
  dblcomplex *buffer=globalbuffer+bands*bands*globalindex,
    *cldos=globalcldos+bands*bands*globalindex;
  gpuint ipos=(gpuint)i-n2, jpos=(gpuint)j-n2;
  for(gpuuint ii=0;ii<oversamp;ii++)
    for(gpuuint jj=0;jj<oversamp;jj++)
      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=0.0f;
  //R-loop over nearest neighbours
  for(gpuint nni1=-window;nni1<=(gpuint)window;nni1++)
    for(gpuint nnj1=-window;nnj1<=(gpuint)window;nnj1++) {
      //calculate TG0(0,R)
      matrixmultiplication(scat,g0+IDX4C((kpoints+kpoints-(ipos+nni1))%kpoints,(kpoints+kpoints-(jpos+nnj1))%kpoints,0,0,kpoints,bands),buffer,bands);
      //R'-loop over nearest neighbours
      for(gpuint nni2=-window;nni2<=(gpuint)window;nni2++)
	for(gpuint nnj2=-window;nnj2<=(gpuint)window;nnj2++) {
	  //calculate G0(R',0)TG0(0,R)    
	    matrixmultiplication(g0+IDX4C((kpoints+ipos+nni2)%kpoints,(kpoints+jpos+nnj2)%kpoints,0,0,kpoints,bands),buffer,cldos,bands);	      
	  //sum over sub-unit cell sampling
	  //gpuuint x=ipos*oversamp+ii,y=jpos*oversamp+jj;
	  for(gpuuint ii=0;ii<oversamp;ii++)
	    for(gpuuint jj=0;jj<oversamp;jj++) {
	      gpuuint x=ii+window*oversamp,y=jj+window*oversamp;
	      gpufloat val=*(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern));
	      for(gpuuint o1=0;o1<maxband;o1++)
		for(gpuuint o2=0;o2<maxband;o2++) {
		  gpufloat factor=*(wf+IDX3C(o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  dblcomplex cval=*(cldos+IDX2C(o2,o1+maxband,bands))+*(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,o2,o1+maxband,kpoints,bands));
		  val-=(cval*tip).imag*factor;
		}
	      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=val;
	    }
	}
    }
}

//calculate LDOS using Wannier functions - using cached functions
//g0 - two-dimensional array of complex matrices (kpointsxkpointsxbandsxbands)
//ldos - two dimensioanl array of double values (nxn)
//scat - matrix (bandsxbands)
//wf - array of two-dimensional arrays containing orbitals (bandsx((2*windows+1)*oversamp)x((2*windows+1)*oversamp))
//kpoints - number of kpoints, as specified in g0
//n - number of lattice sites in real-space lattice
//bands - number of bands
//oversamp - oversampling of real-space lattice
//window - window used for orbitals
//maxband - band up to which the orbitals are included (e.g. for superconducting calculation)
//spin - spin-polarized calculation
__global__ void gpucalcwannierjosephsonspin(const dblcomplex *g0, gpufloat *ldos,const dblcomplex *scat,const gpufloat *wf,dblcomplex *globalbuffer,dblcomplex *globalcldos,const dblcomplex tip,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint oversamp,gpuuint window,gpuuint maxband) {
  gpuuint wanniern=oversamp*n;
  gpuuint n2=(n>>1);
  gpuuint wfsize=(2*window+1)*oversamp;
  gpuuint spinbands=maxband>>1;
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  //sum over lattice
  dblcomplex *buffer=globalbuffer+bands*bands*globalindex,
    *cldos=globalcldos+bands*bands*globalindex;
  gpuint ipos=(gpuint)i-n2, jpos=(gpuint)j-n2;
  for(gpuuint ii=0;ii<oversamp;ii++)
    for(gpuuint jj=0;jj<oversamp;jj++)
      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=0.0f;
  //R-loop over nearest neighbours
  for(gpuint nni1=-window;nni1<=(gpuint)window;nni1++)
    for(gpuint nnj1=-window;nnj1<=(gpuint)window;nnj1++) {
      //calculate TG0(0,R)
      matrixmultiplication(scat,g0+IDX4C((kpoints+kpoints-(ipos+nni1))%kpoints,(kpoints+kpoints-(jpos+nnj1))%kpoints,0,0,kpoints,bands),buffer,bands);
      //R'-loop over nearest neighbours
      for(gpuint nni2=-window;nni2<=(gpuint)window;nni2++)
	for(gpuint nnj2=-window;nnj2<=(gpuint)window;nnj2++) {
	  //calculate G0(R',0)TG0(0,R)    
	    matrixmultiplication(g0+IDX4C((kpoints+ipos+nni2)%kpoints,(kpoints+jpos+nnj2)%kpoints,0,0,kpoints,bands),buffer,cldos,bands);
	  //gpuuint x=ipos*oversamp+ii,y=jpos*oversamp+jj;
	  //sum over sub-unit cell sampling
	  for(gpuuint ii=0;ii<oversamp;ii++)
	    for(gpuuint jj=0;jj<oversamp;jj++) {
	      gpuuint x=ii+window*oversamp,y=jj+window*oversamp;
	      gpufloat val=*(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern));
	      for(gpuuint o1=0;o1<spinbands;o1++)
		for(gpuuint o2=0;o2<spinbands;o2++) {
		  gpufloat factor=*(wf+IDX3C(o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  dblcomplex cval=*(cldos+IDX2C(o2,o1+maxband,bands))+*(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,o2,o1+maxband,kpoints,bands));
		  val-=(cval*tip).imag*factor;
		  factor=*(wf+IDX3C(spinbands+o1,x-nni1*oversamp,y-nnj1*oversamp,wfsize))*(*(wf+IDX3C(spinbands+o2,x-nni2*oversamp,y-nnj2*oversamp,wfsize)));
		  cval=*(cldos+IDX2C(spinbands+o2,spinbands+o1+maxband,bands))+*(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,spinbands+o2,spinbands+o1+maxband,kpoints,bands));
		  val-=(cval*tip).imag*factor;
		}
	      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=val;
	    }
	}
    }
}

//calculate LDOS using Wannier functions - using cached functions
//kernel void gpucalcwannierldos(device const dblcomplex *g0, device gpufloat *ldos,device const dblcomplex *scat,device const gpufloat *wf,device dblcomplex *gblbuffer,device dblcomplex *gblcldos,constant const qpigpuinfo *qpiinfoblock, uint globalindex [[thread_position_in_grid]])
__global__ void gpucalcwannierldoslist(const dblcomplex *g0, gpufloat *ldos,const dblcomplex *scat,const qpigpuflaglist *flags,const gpuuint *flagoffsets,const gpuuint *flagentries,dblcomplex *gblbuffer,dblcomplex *gblcldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint oversamp,gpuuint window) {
  gpuuint wanniern=oversamp*n;
  gpuuint n2=(n>>1);
  gpuuint winn=(2*window+1);
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  dblcomplex *buffer=gblbuffer+bands*bands*globalindex,
    *cldos=gblcldos+bands*bands*globalindex;
  if(j>=n) return;
  gpuint ipos=(gpuint)i-n2, jpos=(gpuint)j-n2;
  for(gpuuint ii=0;ii<oversamp;ii++)
    for(gpuuint jj=0;jj<oversamp;jj++)
      *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=0.0;
  //R-loop over nearest neighbours
  for(gpuint nni1=-window;nni1<=(gpuint)window;nni1++)
    for(gpuint nnj1=-window;nnj1<=(gpuint)window;nnj1++) {
      //calculate TG0(0,R)
      matrixmultiplication(scat,g0+IDX4C((kpoints+kpoints-(ipos+nni1))%kpoints,(kpoints+kpoints-(jpos+nnj1))%kpoints,0,0,kpoints,bands),buffer,bands);
      //matrixmultiplication(scat,g0[(kpoints+kpoints-(ipos+nni1))%kpoints][(kpoints+kpoints-(jpos+nnj1))%kpoints],buffer,bands);
      //R'-loop over nearest neighbours
      for(gpuint nni2=-window;nni2<=(gpuint)window;nni2++)
	for(gpuint nnj2=-window;nnj2<=(gpuint)window;nnj2++) {
	  //calculate G0(R',0)TG0(0,R)
	  matrixmultiplication(g0+IDX4C((kpoints+ipos+nni2)%kpoints,(kpoints+jpos+nnj2)%kpoints,0,0,kpoints,bands),buffer,cldos,bands);
	  //matrixmultiplication(g0[(kpoints+ipos+nni2)%kpoints][(kpoints+jpos+nnj2)%kpoints],buffer,cldos,bands);
	  //sum over sub-unit cell sampling
	  gpuuint listlen=*(flagentries+IDX4CC(nni1+window,nnj1+window,nni2+window,nnj2+window,winn));
	  //flags[nni1+window][nnj1+window][nni2+window][nnj2+window].size();
	  gpuuint listofs=*(flagoffsets+IDX4CC(nni1+window,nnj1+window,nni2+window,nnj2+window,winn));
	  for(gpuuint ind=0;ind<listlen;ind++) {
	    gpuuint ii=(flags+listofs+ind)->i,
	      jj=(flags+listofs+ind)->j,
	      o1=(flags+listofs+ind)->o1,
	      o2=(flags+listofs+ind)->o2;
	    gpufloat factor=(flags+listofs+ind)->factor;
	    gpufloat val=*(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern));
	    val-=(cldos+IDX2C(o2,o1,bands))->imag*factor;
	    val-=(g0+IDX4C((kpoints-nni1+nni2)%kpoints,(kpoints-nnj1+nnj2)%kpoints,o2,o1,kpoints,bands))->imag*factor;
	    *(ldos+IDX2C((i*oversamp+ii)%wanniern,(j*oversamp+jj)%wanniern,wanniern))=val;
	  }
	}
    }
}

//calculate spectral function
__global__ void gpucalcspf(const dblcomplex *g0, gpufloat *ldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint maxband) {
  gpuuint n2=(n>>1); //this fixes the centre being in the wrong position
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  gpufloat trace=0.0;
  for(gpuuint k=0;k<maxband;k++)
    //calculate G0(0,R')=G0(0,R')
    trace-=(g0+IDX4C(i,j,k,k,kpoints,bands))->imag;
  *(ldos+IDX2C((n+i-n2)%n,(n+j-n2)%n,n))=trace;
}

//calculate unfolded spectral function
__global__ void gpucalcuspf(const dblcomplex *g0, const gpufloat *pos,gpufloat *ldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint maxband) {
  gpuuint n2=(n>>1); //this fixes the centre being in the wrong position
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  gpufloat trace=0.0;
  gpufloat kx=((gpufloat)i-n2)/(gpufloat)kpoints,ky=((gpufloat)j-n2)/(gpufloat)kpoints;
  gpuint ki=((i+n2)%n+kpoints)%kpoints,kj=((j+n2)%n+kpoints)%kpoints;
  for(gpuuint k=0;k<maxband;k++)
    for(gpuuint l=0;l<maxband;l++) {
      //calculate G0(0,R')=G0(0,R')
      dblcomplex prefact=cmpexp((dblcomplex){0.0,2.0*HIP_PI*(kx*(*(pos+IDX2C(0,k,3))-*(pos+IDX2C(0,l,3)))+ky*(*(pos+IDX2C(1,k,3))-*(pos+IDX2C(1,l,3))))});
      gpufloat flpref=(*(pos+IDX2C(2,k,3)))*(*(pos+IDX2C(2,l,3)));
      trace-=(*(g0+IDX4C(ki,kj,l,k,kpoints,bands))*prefact).imag*flpref;
    }
  *(ldos+IDX2C(i,j,n))=trace;
}

//calculate unfolded spectral function
__global__ void gpucalcuspfspin(const dblcomplex *g0, const gpufloat *pos,gpufloat *ldos,gpuuint kpoints,gpuuint n,gpuuint bands,gpuuint maxband) {
  gpuuint n2=(n>>1); //this fixes the centre being in the wrong position
  gpuuint spinbands=maxband>>1;
  gpuuint globalindex=blockIdx.x * blockDim.x + threadIdx.x;
  gpuuint i=globalindex%n,j=globalindex/n;
  if(j>=n) return;
  gpufloat trace=0.0;
  gpufloat kx=((gpufloat)i-n2)/(gpufloat)kpoints,ky=((gpufloat)j-n2)/(gpufloat)kpoints;
  gpuint ki=((i+n2)%n+kpoints)%kpoints,kj=((j+n2)%n+kpoints)%kpoints;
  for(gpuuint k=0;k<spinbands;k++)
    for(gpuuint l=0;l<spinbands;l++) {
      //calculate G0(0,R')=G0(0,R')
      dblcomplex prefact=cmpexp((dblcomplex){0.0,2.0*HIP_PI*(kx*(*(pos+IDX2C(0,k,3))-*(pos+IDX2C(0,l,3)))+ky*(*(pos+IDX2C(1,k,3))-*(pos+IDX2C(1,l,3))))});
      gpufloat flpref=(*(pos+IDX2C(2,k,3)))*(*(pos+IDX2C(2,l,3)));
      trace-=(*(g0+IDX4C(ki,kj,l,k,kpoints,bands))*prefact).imag*flpref;
      prefact=cmpexp((dblcomplex){0.0,2.0*HIP_PI*(kx*(*(pos+IDX2C(0,spinbands+k,3))-*(pos+IDX2C(0,spinbands+l,3)))+ky*(*(pos+IDX2C(1,spinbands+k,3))-*(pos+IDX2C(1,spinbands+l,3))))});
      flpref=(*(pos+IDX2C(2,spinbands+k,3)))*(*(pos+IDX2C(2,spinbands+l,3)));
      trace-=(*(g0+IDX4C(ki,kj,spinbands+l,spinbands+k,kpoints,bands))*prefact).imag*flpref;
    }
  *(ldos+IDX2C(i,j,n))=trace;
}

#define gpuErrchk(ostr,ans) { gpuAssert((ostr),(ans), __FILE__, __LINE__); }

inline void gpuAssert(ostream &os,hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    os<<"CPU CUDA error: "<<hipGetErrorString(code)<<" thrown in "<<file<<", l."<<line<<endl;
    if (abort) exit(code);
  }
}

//note: nkpts needs to be at least (n+1)
void CudaQPI::copygf(vector<vector<gsl_matrix_complex *> > &g0) {
  int n2=nkpts>>1;
#pragma omp parallel for
  for(int i=-n2;i<n2;i++)
    for(int j=-n2;j<n2;j++)
      for(size_t k=0;k<bands;k++)
	for(size_t l=0;l<bands;l++) {
	  gsl_complex c=gsl_matrix_complex_get(g0[(kpoints+i)%kpoints][(kpoints+j)%kpoints],k,l);
	  *(gpug0mem+IDX4C((nkpts+i)%nkpts,(nkpts+j)%nkpts,k,l,nkpts,bands))=(dblcomplex){(gpufloat)GSL_REAL(c),(gpufloat)GSL_IMAG(c)};
	}
}

CudaQPI::CudaQPI(size_t wanniern,size_t kpoints,size_t n,size_t window,size_t oversamp,size_t bands,size_t maxband,bool spin,vector<wannierfunctions> &wf):wanniern(wanniern),kpoints(kpoints),n(n),window(window),oversamp(oversamp),bands(bands),maxband(maxband),spin(spin) {
#ifdef _mpi_version
  if(world_size>1) {
    int count;
    hipGetDeviceCount(&count);
    hipSetDevice(world_rank%count);
  }
#endif
  gpumem=0; cpumem=0; nkpts=n+2*window+2;
  scatsize=sizeof(dblcomplex)*bands*bands; g0size=sizeof(dblcomplex)*bands*bands*nkpts*nkpts; ldossize=wanniern*wanniern*sizeof(gpufloat);
  size_t wfsize=maxband*(2*window+1)*oversamp*(2*window+1)*oversamp*sizeof(gpufloat),bufsize=n*n*bands*bands*sizeof(dblcomplex);
  gpuscat=new dblcomplex[bands*bands]; cpumem+=bands*bands*sizeof(dblcomplex);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuscat, scatsize)); gpumem+=scatsize;
  gpug0mem=new dblcomplex[bands*bands*nkpts*nkpts]; cpumem+=bands*bands*nkpts*nkpts*sizeof(dblcomplex);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpug0mem, g0size)); gpumem+=g0size;
  gpufloat *gpuwf=new gpufloat[bands*(2*window+1)*oversamp*(2*window+1)*oversamp];
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuwf, wfsize)); gpumem+=wfsize;
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuldosmem, ldossize)); gpumem+=ldossize;
  gpuldosmem=NULL;
  for(size_t i=0;i<maxband;i++)
    for(size_t j=0;j<(2*window+1)*oversamp;j++)
      for(size_t k=0;k<(2*window+1)*oversamp;k++)
	*(gpuwf+IDX3C(i,j,k,(2*window+1)*oversamp))=wf[i].getwave_cached(j,k);
  hipMemcpy(d_gpuwf, gpuwf, wfsize, hipMemcpyHostToDevice);
  free(gpuwf);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpubuf,bufsize)); gpumem+=bufsize;
  gpuErrchk(cerr,hipMalloc((void **)&d_gpucldos,bufsize)); gpumem+=bufsize;
  //hipDeviceSetLimit(hipLimitMallocHeapSize,);
  //setting parameters for GPU  
  size_t total=n*n;
  ldosblocksize=512;
  ldosblocks=total/ldosblocksize;
  if(total%ldosblocksize) ldosblocks++;
  d_gpucontg=NULL; d_mpos=NULL;
}

CudaQPI::CudaQPI(size_t wanniern, size_t kpoints,size_t n,size_t window, size_t oversamp,size_t bands,flaglist &flist):wanniern(wanniern),kpoints(kpoints),n(n),window(window),oversamp(oversamp),bands(bands) {
#ifdef _mpi_version
  if(world_size>1) {
    int count;
    hipGetDeviceCount(&count);
    hipSetDevice(world_rank%count);
  }
#endif
  gpumem=0; cpumem=0; nkpts=n+2*window+2;
  size_t winn=2*window+1,nwinn=winn*winn,nwinn2=nwinn*nwinn,tflentries=0;
  scatsize=sizeof(dblcomplex)*bands*bands; g0size=sizeof(dblcomplex)*bands*bands*nkpts*nkpts; ldossize=wanniern*wanniern*sizeof(gpufloat);
  size_t bufsize=n*n*bands*bands*sizeof(dblcomplex);
  gpuscat=new dblcomplex[bands*bands]; cpumem+=bands*bands*sizeof(dblcomplex);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuscat, scatsize)); gpumem+=scatsize;
  gpug0mem=new dblcomplex[bands*bands*nkpts*nkpts]; cpumem+=bands*bands*nkpts*nkpts*sizeof(dblcomplex);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpug0mem, g0size)); gpumem+=g0size;
  d_gpuwf=NULL;
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuldosmem, ldossize)); gpumem+=ldossize;
  gpuldosmem=NULL;
  gpuuint *flgentr,*flgofs;
  size_t flagarrsize=nwinn2*sizeof(gpuuint);
  qpigpuflaglist *gpuflg;
  flgofs=new gpuuint[nwinn2];
  gpuErrchk(cerr,hipMalloc((void **)&d_flagofs, flagarrsize)); gpumem+=flagarrsize;
  flgentr=new gpuuint[nwinn2];
  gpuErrchk(cerr,hipMalloc((void **)&d_flagentries, flagarrsize)); gpumem+=flagarrsize;
  for(size_t i=0;i<winn;i++)
    for(size_t j=0;j<winn;j++)
      for(size_t k=0;k<winn;k++)
	for(size_t l=0;l<winn;l++) {
	  size_t ofs=IDX4CC(i,j,k,l,winn);
	  *(flgofs+ofs)=tflentries;
	  *(flgentr+ofs)=flist[i][j][k][l].size();
	  tflentries+=flist[i][j][k][l].size();
	}
  size_t flsize=tflentries*sizeof(qpigpuflaglist);
  gpuflg=new qpigpuflaglist[tflentries];
  gpuErrchk(cerr,hipMalloc((void **)&d_flaglist, flsize)); gpumem+=flsize;
  for(size_t i=0;i<winn;i++)
    for(size_t j=0;j<winn;j++)
      for(size_t k=0;k<winn;k++)
	for(size_t l=0;l<winn;l++) {
	  size_t ofs=IDX4CC(i,j,k,l,winn),
	    flpos=*(flgofs+ofs),
	    fllen=*(flgentr+ofs);
	  for(size_t m=0;m<fllen;m++) {
	    (gpuflg+flpos+m)->i=flist[i][j][k][l][m].i;
	    (gpuflg+flpos+m)->j=flist[i][j][k][l][m].j;
	    (gpuflg+flpos+m)->o1=flist[i][j][k][l][m].o1;
	    (gpuflg+flpos+m)->o2=flist[i][j][k][l][m].o2;
	    (gpuflg+flpos+m)->factor=flist[i][j][k][l][m].factor;
	  }
	}
  hipMemcpy(d_flagofs, flgofs, flagarrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_flagentries, flgentr, flagarrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_flaglist, gpuflg, flsize, hipMemcpyHostToDevice);
  free(gpuflg); free(flgofs); free(flgentr);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpubuf,bufsize)); gpumem+=bufsize;
  gpuErrchk(cerr,hipMalloc((void **)&d_gpucldos,bufsize)); gpumem+=bufsize;
  //hipDeviceSetLimit(hipLimitMallocHeapSize,);
  //setting parameters for GPU  
  size_t total=n*n;
  ldosblocksize=512;
  ldosblocks=total/ldosblocksize;
  if(total%ldosblocksize) ldosblocks++;
  d_gpucontg=NULL; d_mpos=NULL;
}

CudaQPI::CudaQPI(size_t kpoints,size_t n,size_t bands,size_t maxbands,bool spin,vector<vector<double> > &pos,vector<double> &prearr):kpoints(kpoints),n(n),wanniern(n),bands(bands),maxband(maxbands),spin(spin) {
#ifdef _mpi_version
  if(world_size>1) {
    int count;
    hipGetDeviceCount(&count);
    hipSetDevice(world_rank%count);
  }
#endif
  gpumem=0; cpumem=0; nkpts=kpoints;
  scatsize=0; g0size=sizeof(dblcomplex)*bands*bands*nkpts*nkpts; ldossize=wanniern*wanniern*sizeof(gpufloat);
  gpug0mem=new dblcomplex[bands*bands*nkpts*nkpts]; cpumem+=bands*bands*nkpts*nkpts*sizeof(dblcomplex);
  gpuErrchk(cerr,hipMalloc((void **)&d_gpug0mem, g0size)); gpumem+=g0size;
  gpuErrchk(cerr,hipMalloc((void **)&d_gpuldosmem, ldossize)); gpumem+=ldossize;
  gpuscat=NULL; gpuldosmem=NULL; d_gpuscat=NULL; d_gpubuf=NULL; d_gpucldos=NULL; d_gpucontg=NULL;
  if(pos.size()) {
    size_t possize=pos.size()*3*sizeof(gpufloat);
    gpuErrchk(cerr,hipMalloc((void **)&d_mpos, possize)); gpumem+=possize;
    gpufloat *gpupos= new gpufloat[3*pos.size()];
    for(size_t i=0;i<pos.size();i++) {
      for(size_t j=0;j<2;j++)
	*(gpupos+IDX2C(j,i,3))=pos[i][j];
      *(gpupos+IDX2C(2,i,3))=prearr[i%prearr.size()];
    }
    hipMemcpy(d_mpos, gpupos, possize, hipMemcpyHostToDevice);
    free(gpupos);
  }
  
  //hipDeviceSetLimit(hipLimitMallocHeapSize,);
  //setting parameters for GPU  
  size_t total=n*n;
  ldosblocksize=512;
  ldosblocks=total/ldosblocksize;
  if(total%ldosblocksize) ldosblocks++;
  d_gpucontg=NULL;
}

void CudaQPI::printinfo(ostream &os) {
  int numdevices;
  hipGetDeviceCount(&numdevices);
  if(numdevices==0)
    os<<"Error: no CUDA GPUs found."<<endl;
  else
    os<<"Running continuum QPI on Cuda GPU."<<endl;
  for(int device=0;device<numdevices;device++) {
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
#ifdef _mpi_version
    if(world_size>1) {
      os<<"MPI Task "<<world_rank<<"/"<<world_size<<": "<<endl;
      if((world_rank%numdevices)==device) os<<"CUDA GPU "<<device<<"/"<<numdevices<<": "<<props.name<<endl;
    } else
#endif
      os<<"CUDA GPU "<<device<<"/"<<numdevices<<": "<<props.name<<endl;
    ExecuteCPU0 {
      os<<"Additional memory requirements:"<<endl
	<<"GPU memory:       "<<std::fixed<<std::setprecision(2)<<(double)gpumem/1024.0/1024.0/1024.0<<"GB"<<endl
	<<"CPU memory:       "<<std::fixed<<std::setprecision(2)<<(double)cpumem/1024.0/1024.0/1024.0<<"GB"<<endl
	<<"Blocksize:        "<<ldosblocksize<<endl
	<<"Number of blocks: "<<ldosblocks<<endl;
    }
  }
}

CudaQPI::~CudaQPI() {
  free(gpug0mem); hipFree(d_gpug0mem); hipFree(d_gpuldosmem);
  if(gpuldosmem) free(gpuldosmem);
  if(gpuscat) free(gpuscat);
  if(d_gpuscat) hipFree(d_gpuscat);
  if(d_gpubuf) hipFree(d_gpubuf);
  if(d_gpucldos) hipFree(d_gpucldos);
  if(d_gpucontg) hipFree(d_gpucontg);
  if(d_gpuwf) hipFree(d_gpuwf);
  if(d_flaglist) hipFree(d_flaglist);
  if(d_flagentries) hipFree(d_flagentries);
  if(d_flagofs) hipFree(d_flagofs);
  if(d_mpos) hipFree(d_mpos);
}

void CudaQPI::wannierldos(gsl_matrix_complex *scat, vector<vector<gsl_matrix_complex *> > &g0)
{
  //scattering matrix
  for(size_t i=0;i<bands;i++)
    for(size_t j=0;j<bands;j++) {
      gsl_complex c=gsl_matrix_complex_get(scat,i,j);
      *(gpuscat+IDX2C(i,j,bands))=(dblcomplex){(gpufloat)GSL_REAL(c),(gpufloat)GSL_IMAG(c)};
    }
  hipMemcpy(d_gpuscat, gpuscat, scatsize, hipMemcpyHostToDevice);
  copygf(g0);
  hipMemcpy(d_gpug0mem, gpug0mem, g0size, hipMemcpyHostToDevice);
  // cleartotal=wanniern*wanniern;
  // clearblocksize=512;
  // clearblocks=cleartotal/clearblocksize;
  // if(cleartotal%clearblocksize) clearblocks++;
  //clearldos<<<clearblocks,clearblocksize>>>(d_gpuldosmem,cleartotal);
  //hipDeviceSynchronize();
  if(spin)
    gpucalcwannierldosspin<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,d_gpuscat,d_gpuwf,d_gpubuf,d_gpucldos,nkpts,n,bands,oversamp,window,maxband);
  else
    gpucalcwannierldos<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,d_gpuscat,d_gpuwf,d_gpubuf,d_gpucldos,nkpts,n,bands,oversamp,window,maxband);
}

void CudaQPI::wannierjosephson(gsl_matrix_complex *scat, vector<vector<gsl_matrix_complex *> > &g0,gsl_complex tip)
{
  //scattering matrix
  for(size_t i=0;i<bands;i++)
    for(size_t j=0;j<bands;j++) {
      gsl_complex c=gsl_matrix_complex_get(scat,i,j);
      *(gpuscat+IDX2C(i,j,bands))=(dblcomplex){(gpufloat)GSL_REAL(c),(gpufloat)GSL_IMAG(c)};
    }
  hipMemcpy(d_gpuscat, gpuscat, scatsize, hipMemcpyHostToDevice);
  copygf(g0);
  hipMemcpy(d_gpug0mem, gpug0mem, g0size, hipMemcpyHostToDevice);
  // cleartotal=wanniern*wanniern;
  // clearblocksize=512;
  // clearblocks=cleartotal/clearblocksize;
  // if(cleartotal%clearblocksize) clearblocks++;
  //clearldos<<<clearblocks,clearblocksize>>>(d_gpuldosmem,cleartotal);
  //hipDeviceSynchronize();
  dblcomplex ctip=(dblcomplex){(gpufloat)GSL_REAL(tip),(gpufloat)GSL_IMAG(tip)};
  if(spin)
    gpucalcwannierjosephsonspin<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,d_gpuscat,d_gpuwf,d_gpubuf,d_gpucldos,ctip,nkpts,n,bands,oversamp,window,maxband);
  else
    gpucalcwannierjosephson<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,d_gpuscat,d_gpuwf,d_gpubuf,d_gpucldos,ctip,nkpts,n,bands,oversamp,window,maxband);
}

void CudaQPI::wannierldoslist(gsl_matrix_complex *scat, vector<vector<gsl_matrix_complex *> > &g0)
{
  //scattering matrix
  for(size_t i=0;i<bands;i++)
    for(size_t j=0;j<bands;j++) {
      gsl_complex c=gsl_matrix_complex_get(scat,i,j);
      *(gpuscat+IDX2C(i,j,bands))=(dblcomplex){(gpufloat)GSL_REAL(c),(gpufloat)GSL_IMAG(c)};
    }
  hipMemcpy(d_gpuscat, gpuscat, scatsize, hipMemcpyHostToDevice);
  copygf(g0);
  hipMemcpy(d_gpug0mem, gpug0mem, g0size, hipMemcpyHostToDevice);
  gpucalcwannierldoslist<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,d_gpuscat,d_flaglist,d_flagofs,d_flagentries,d_gpubuf,d_gpucldos,nkpts,n,bands,oversamp,window);
}

void CudaQPI::spf(vector<vector<gsl_matrix_complex *> > &g0)
{
  copygf(g0);
  hipMemcpy(d_gpug0mem, gpug0mem, g0size, hipMemcpyHostToDevice);
  gpucalcspf<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_gpuldosmem,nkpts,n,bands,maxband);
}

void CudaQPI::uspf(vector<vector<gsl_matrix_complex *> > &g0)
{
  copygf(g0);
  hipMemcpy(d_gpug0mem, gpug0mem, g0size, hipMemcpyHostToDevice);
  if(spin)
    gpucalcuspfspin<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_mpos,d_gpuldosmem,nkpts,n,bands,maxband);
  else
    gpucalcuspf<<<ldosblocks,ldosblocksize>>>(d_gpug0mem,d_mpos,d_gpuldosmem,nkpts,n,bands,maxband);
}

void CudaQPI::retrieveResult(vector<vector<double> > &ldos)
{
  if(!gpuldosmem) {
    gpuldosmem=new gpufloat[wanniern*wanniern];
    cpumem+=wanniern*wanniern*sizeof(gpufloat);
  }
  hipDeviceSynchronize();
  hipMemcpy(gpuldosmem, d_gpuldosmem, ldossize, hipMemcpyDeviceToHost);
  ldos.resize(wanniern);
#pragma omp parallel
  {
#pragma omp for
    for(size_t i=0;i<wanniern;i++)
      ldos[i].resize(wanniern);
#pragma omp for
    for(size_t i=0;i<wanniern;i++)
      for(size_t j=0;j<wanniern;j++)
	ldos[i][j]=*(gpuldosmem+IDX2C(i,j,wanniern));
  }
}

void CudaQPI::retrieveResult(idl &map,size_t layer)
{
  if(!gpuldosmem) {
    gpuldosmem=new gpufloat[wanniern*wanniern];
    cpumem+=wanniern*wanniern*sizeof(gpufloat);
  }
  hipDeviceSynchronize();
  hipMemcpy(gpuldosmem, d_gpuldosmem, ldossize, hipMemcpyDeviceToHost);
#pragma omp parallel for
  for(size_t i=0;i<wanniern;i++)
    for(size_t j=0;j<wanniern;j++)
      map.set(i,j,layer,*(gpuldosmem+IDX2C(i,j,wanniern))); 
}

void CudaQPI::retrieveResult(double *ldos)
{
  hipDeviceSynchronize();
  if(typeid(gpufloat)!=typeid(double)) {
    if(!gpuldosmem) {
      gpuldosmem=new gpufloat[wanniern*wanniern];
      cpumem+=wanniern*wanniern*sizeof(gpufloat);
    }
    hipMemcpy(gpuldosmem, d_gpuldosmem, ldossize, hipMemcpyDeviceToHost);
#pragma omp parallel for
    for(size_t i=0;i<wanniern*wanniern;i++)
      *(ldos+i)=*(gpuldosmem+i);
  } else
    hipMemcpy(ldos, d_gpuldosmem, ldossize, hipMemcpyDeviceToHost);
}
